#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <iostream>

using namespace nvcuda;

const int WIDTH = 64;
const int HEIGHT = 4096;

// CUDA核函数，使用wmma进行矩阵加法
__global__ void add_images(const half* base_image, const half* noise_image, half* output_image) {
    // 定义wmma矩阵片段
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag;

    // 加载矩阵片段
    wmma::load_matrix_sync(a_frag, base_image, WIDTH);
    wmma::load_matrix_sync(b_frag, noise_image, WIDTH);
    wmma::fill_fragment(c_frag, 0.0f);

    // 进行矩阵加法
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    // 将结果存储回全局内存
    wmma::store_matrix_sync(output_image, c_frag, WIDTH, wmma::mem_row_major);
}

int main() {
    // 分配主机和设备内存
    half* h_base_image;
    half* h_noise_image;
    half* h_output_image;

    half* d_base_image;
    half* d_noise_image;
    half* d_output_image;

    size_t image_size = WIDTH * HEIGHT * sizeof(half);

    hipHostMalloc(&h_base_image, image_size, hipHostMallocDefault);
    hipHostMalloc(&h_noise_image, image_size, hipHostMallocDefault);
    hipHostMalloc(&h_output_image, image_size, hipHostMallocDefault);

    hipMalloc(&d_base_image, image_size);
    hipMalloc(&d_noise_image, image_size);
    hipMalloc(&d_output_image, image_size);

    // 初始化图像数据
    for (int i = 0; i < WIDTH * HEIGHT; ++i) {
        h_base_image[i] = __float2half(static_cast<float>(rand()) / RAND_MAX);
        h_noise_image[i] = __float2half(static_cast<float>(rand()) / RAND_MAX);
    }

    // 复制数据到设备
    hipMemcpy(d_base_image, h_base_image, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_noise_image, h_noise_image, image_size, hipMemcpyHostToDevice);

    // 创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 启动CUDA事件记录
    hipEventRecord(start);

    // 启动CUDA核函数
    dim3 blockDim(16, 16);
    dim3 gridDim((WIDTH + blockDim.x - 1) / blockDim.x, (HEIGHT + blockDim.y - 1) / blockDim.y);
    add_images<<<gridDim, blockDim>>>(d_base_image, d_noise_image, d_output_image);

    // 停止CUDA事件记录
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 计算时间差
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 计算每秒帧数
    float fps = 1000.0f / milliseconds; // 每秒处理的帧数
    std::cout << "Processed at " << fps << " frames per second." << std::endl;

    // 清理内存
    hipHostFree(h_base_image);
    hipHostFree(h_noise_image);
    hipHostFree(h_output_image);
    hipFree(d_base_image);
    hipFree(d_noise_image);
    hipFree(d_output_image);

    // 销毁CUDA事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
