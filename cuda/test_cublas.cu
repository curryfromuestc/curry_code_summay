#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

const int WIDTH = 64;
const int HEIGHT = 4096;

// 检查CUDA错误
#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

// 检查cuBLAS错误
#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << std::endl; \
            std::exit(EXIT_FAILURE); \
        } \
    } while (0)

int main() {
    float* h_base_image;
    float* h_noise_image;
    float* h_output_image;

    float* d_base_image;
    float* d_noise_image;
    float* d_output_image;

    size_t image_size = WIDTH * HEIGHT * sizeof(float);

    // 分配主机内存
    CHECK_CUDA(hipHostMalloc(&h_base_image, image_size, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_noise_image, image_size, hipHostMallocDefault));
    CHECK_CUDA(hipHostMalloc(&h_output_image, image_size, hipHostMallocDefault));

    // 分配设备内存
    CHECK_CUDA(hipMalloc(&d_base_image, image_size));
    CHECK_CUDA(hipMalloc(&d_noise_image, image_size));
    CHECK_CUDA(hipMalloc(&d_output_image, image_size));

    // 初始化图像数据
    for (int i = 0; i < WIDTH * HEIGHT; ++i) {
        h_base_image[i] = static_cast<float>(rand()) / RAND_MAX;
        h_noise_image[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 复制数据到设备
    CHECK_CUDA(hipMemcpy(d_base_image, h_base_image, image_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_noise_image, h_noise_image, image_size, hipMemcpyHostToDevice));

    // 创建cuBLAS句柄
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // 创建CUDA事件
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // 设置加法系数
    const float alpha = 1.0f;

    // 记录开始事件
    CHECK_CUDA(hipEventRecord(start, 0));

    // 使用cuBLAS进行矢量加法 y = alpha * x + y (base_image + noise_image)
    CHECK_CUBLAS(hipblasSaxpy(handle, WIDTH * HEIGHT, &alpha, d_base_image, 1, d_noise_image, 1));

    // 记录结束事件
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    // 计算执行时间
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    float fps = 1000.0f / milliseconds;

    std::cout << "Processed at " << fps << " frames per second." << std::endl;

    // 复制结果回主机
    CHECK_CUDA(hipMemcpy(h_output_image, d_noise_image, image_size, hipMemcpyDeviceToHost));

    // 清理资源
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipHostFree(h_base_image));
    CHECK_CUDA(hipHostFree(h_noise_image));
    CHECK_CUDA(hipHostFree(h_output_image));
    CHECK_CUDA(hipFree(d_base_image));
    CHECK_CUDA(hipFree(d_noise_image));
    CHECK_CUDA(hipFree(d_output_image));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return 0;
}
